#include "hip/hip_runtime.h"
#include "funcionesGPU.cuh"

void checkCudaError(hipError_t result, const char* msg)
{
    if (result != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " : " << hipGetErrorString(result) << std::endl;
        system("PAUSE");
        exit(result);
    }
}
void instanciarMemGPU(Coord** posAntGPU, Coord** posActGPU, Coord** posSigGPU, float** velocidadesGPU, unsigned int numBytesPos, unsigned int numBytesVel, Coord* partAct)
{
    checkCudaError(hipMalloc(posAntGPU, numBytesPos), "cudaMalloc1 posAntGPU");
    checkCudaError(hipMalloc(posActGPU, numBytesPos), "cudaMalloc2 posActGPU");
    checkCudaError(hipMalloc(posSigGPU, numBytesPos), "cudaMalloc3 posSigGPU");
    checkCudaError(hipMalloc(velocidadesGPU, numBytesVel), "cudaMalloc4 velocidadesGPU");

    checkCudaError(hipMemcpy(*posAntGPU, partAct, numBytesPos, hipMemcpyHostToDevice), "cudaMemcpy1 posAntGPU");
    checkCudaError(hipMemcpy(*posActGPU, partAct, numBytesPos, hipMemcpyHostToDevice), "cudaMemcpy2 posActGPU");
}

extern "C" __global__ void calculoNuevaPosicionGPU(Coord * posAntGPU, Coord * posActGPU, Coord * posSigGPU, int numParticulas, float pasoTiempo, float constGrav)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < numParticulas)
    {
       /*
        __shared Coord particulasBloque[256];

        int inicioBloque = 0;


        particulasBloque[threadIdx.x] = postActGPU[inicioBloque + threadIdx.x]
        _    
        */
        Coord Pi = posActGPU[index];
        Coord PiAnt = posAntGPU[index];
        Coord fuerza = { 0.0f, 0.0f };
        float minDist = 0.0012f; //aqu� tengo que ir ajustando este par�metro. CU�NTO M�S PEQUE�O M�S SE DISPERSA
        
        // Calcular la fuerza gravitacional
        for (int j = 0; j < numParticulas; ++j) {
            if (index != j) {
                Coord Pj = posActGPU[j];
                Coord dirVector = { Pj.x - Pi.x, Pj.y - Pi.y };
                float dist = sqrtf(dirVector.x * dirVector.x + dirVector.y * dirVector.y);
                if (dist > minDist)
                {
                    float distCubica = dist * dist * dist;
                    float factor = constGrav / distCubica;
                    fuerza.x += factor * dirVector.x;
                    fuerza.y += factor * dirVector.y;
                }
            }
        }
        //Integraci�n de Verlet
        posSigGPU[index].x = 2 * Pi.x - PiAnt.x + (pasoTiempo * pasoTiempo) * fuerza.x;
        posSigGPU[index].y = 2 * Pi.y - PiAnt.y + (pasoTiempo * pasoTiempo) * fuerza.y;
    }
}

extern "C" __global__ void calculoVelocidadesGPU(Coord * posAntGPU, Coord * posActGPU, Coord * posSigGPU, float* velocidadesGPU, int numParticulas, float pasoTiempo, float constGrav)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < numParticulas)
    {
        Coord Pi = posActGPU[index];
        Coord PiAnt = posAntGPU[index];
        Coord fuerza = { 0.0f, 0.0f };
        float minDist = 0.002f; //aqu� tengo que ir ajustando este par�metro. CU�NTO M�S PEQUE�O M�S SE DISPERSA
        
        // Calcular la fuerza gravitacional
        for (int j = 0; j < numParticulas; ++j) {
            if (index != j) {
                Coord Pj = posActGPU[j];
                Coord dirVector = { Pj.x - Pi.x, Pj.y - Pi.y };
                float dist = sqrtf(dirVector.x * dirVector.x + dirVector.y * dirVector.y);
                if (dist > minDist)
                {
                    float distCubica = dist * dist * dist;
                    float factor = constGrav / distCubica;
                    fuerza.x += factor * dirVector.x;
                    fuerza.y += factor * dirVector.y;
                }
            }
        }
        //Integraci�n de Verlet
        posSigGPU[index].x = 2 * Pi.x - PiAnt.x + (pasoTiempo * pasoTiempo) * fuerza.x;
        posSigGPU[index].y = 2 * Pi.y - PiAnt.y + (pasoTiempo * pasoTiempo) * fuerza.y;

        //C�lculo de la velocidad de cada part�cula
        float velX = (posSigGPU[index].x - posActGPU[index].x) / pasoTiempo;
        float velY = (posSigGPU[index].y - posActGPU[index].y) / pasoTiempo;
        velocidadesGPU[index] = sqrtf(velX * velX + velY * velY);
    }
}

void lanzarKernelCalculoPosicion(Coord* posAntGPU, Coord* posActGPU, Coord* posSigGPU, int numParticulas, float pasoTiempo, float constGrav, int numBlocks, int blockSize)
{
    calculoNuevaPosicionGPU << <numBlocks, blockSize >> > (posAntGPU, posActGPU, posSigGPU, numParticulas, pasoTiempo, constGrav);
}

void lanzarKernelCalculoVelocidad(Coord* posAntGPU, Coord* posActGPU, Coord* posSigGPU, float* velocidadesGPU, int numParticulas, float pasoTiempo, float constGrav, int numBlocks, int blockSize)
{
    calculoVelocidadesGPU << <numBlocks, blockSize >> > (posAntGPU, posActGPU, posSigGPU, velocidadesGPU, numParticulas, pasoTiempo, constGrav);
}

void obtenerResultadoPosicionGPU(Coord* posAntGPU, Coord* posActGPU, Coord* posSigGPU, int numBytes, Coord* partAct)
{
    hipDeviceSynchronize(); //Para asegurarnos que ya todos los calculos de las hebras de CUDA han terminado

    //Actualizamos los vectores de las posiciones de part�culas dentro de la memoria de la GPU
    checkCudaError(hipMemcpy(posAntGPU, posActGPU, numBytes, hipMemcpyDeviceToDevice), "cudaMemcpy3 posAntGPU");
    //Resultado del calculo se guarda en el vector de particulas ACTUAL
    checkCudaError(hipMemcpy(partAct, posSigGPU, numBytes, hipMemcpyDeviceToHost), "cudaMemcpy4 partAct");
    //Actualizamos los vectores de las posiciones de part�culas dentro de la memoria de la GPU
    checkCudaError(hipMemcpy(posActGPU, posSigGPU, numBytes, hipMemcpyDeviceToDevice), "cudaMemcpy5 posActGPU");

    //En principio el vector de part�culas SIGUIENTE se puede quedar con su contenido actual a modo de basura sin que en principio esto afecte al funcionamiento

}

void obtenerResultadoVelocidadGPU(Coord* posAntGPU, Coord* posActGPU, Coord* posSigGPU, float* velocidadesGPU, float* velocidadesHost, int numBytesParticulas, int numBytesVel)
{
    hipDeviceSynchronize(); //Para asegurarnos que ya todos los calculos de las hebras de CUDA han terminado

    //Actualizamos los vectores de las posiciones de part�culas dentro de la memoria de la GPU
    checkCudaError(hipMemcpy(posAntGPU, posActGPU, numBytesParticulas, hipMemcpyDeviceToDevice), "cudaMemcpy3 posAntGPU");
    //Actualizamos los vectores de las posiciones de part�culas dentro de la memoria de la GPU
    checkCudaError(hipMemcpy(posActGPU, posSigGPU, numBytesParticulas, hipMemcpyDeviceToDevice), "cudaMemcpy4 posActGPU");
    
    //En principio el vector de part�culas SIGUIENTE se puede quedar con su contenido actual a modo de basura sin que en principio esto afecte al funcionamiento
    
    //Actualizamos el vector de velocidades en el host
    checkCudaError(hipMemcpy(velocidadesHost, velocidadesGPU, numBytesVel, hipMemcpyDeviceToHost), "cudaMemcpy5 velocidadesHost");
}

void liberarRecursosGPU(Coord* posAntGPU, Coord* posActGPU, Coord* posSigGPU, float* velocidadesGPU)
{
    hipFree(posAntGPU);
    hipFree(posActGPU);
    hipFree(posSigGPU);
    hipFree(velocidadesGPU);
}


